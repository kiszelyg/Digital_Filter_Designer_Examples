#include "hip/hip_runtime.h"
// iircu_cu.txt template file, version: 01_01_01
// GENERATED FILE! MODIFY THIS FILE ONLY AT YOUR OWN RESPONSIBLITY!
// An identical behaviour to the simulation results can be assured only if this file remains unchanged!
// Code file of a general CUDA (R) IIR filter implementation

#include "iircu.cuh"

template<typename T>
__global__ void fwdFiltering(CIirCu<T>* f_segment_ptr, unsigned short m_numOfStages, unsigned char f_bitShifter, T f_inputVal, unsigned char f_n, T* f_output_ptr)
{
    T f_tempVal = f_inputVal;
    for (unsigned short l_idx = 0; l_idx < m_numOfStages; l_idx++)
    {
        f_segment_ptr[l_idx].m_x = f_tempVal;
        if (0 == f_bitShifter)
        {
            f_segment_ptr[l_idx].m_y = (f_segment_ptr[l_idx].m_coeffs.m_b0 * f_segment_ptr[l_idx].m_x + f_segment_ptr[l_idx].m_t1_vec[(f_n + 1) & 1]);
        }
        else
        {
            f_segment_ptr[l_idx].m_y = ((long long)(f_segment_ptr[l_idx].m_coeffs.m_b0 * f_segment_ptr[l_idx].m_x + f_segment_ptr[l_idx].m_t1_vec[(f_n + 1) & 1]) >> f_bitShifter);
        }
        f_tempVal = f_segment_ptr[l_idx].m_y;
    }
    *f_output_ptr = f_tempVal;
}

template<typename T>
__global__ void rwdFiltering(CIirCu<T>* f_segment_ptr, unsigned char f_n)
{
    const unsigned short l_idx = threadIdx.x;
    T l_xB1 = f_segment_ptr[l_idx].m_coeffs.m_b1 * f_segment_ptr[l_idx].m_x;
    T l_xB2 = f_segment_ptr[l_idx].m_coeffs.m_b2 * f_segment_ptr[l_idx].m_x;
    T l_yA1 = f_segment_ptr[l_idx].m_coeffs.m_a1 * f_segment_ptr[l_idx].m_y;
    T l_yA2 = f_segment_ptr[l_idx].m_coeffs.m_a2 * f_segment_ptr[l_idx].m_y;
    f_segment_ptr[l_idx].m_t1_vec[f_n & 1] = l_xB1 + f_segment_ptr[l_idx].m_t2_vec[(f_n + 1) & 1] + l_yA1;
    f_segment_ptr[l_idx].m_t2_vec[f_n & 1] = l_xB2 + l_yA2;
}

template<typename T>
__global__ void resetSegment(CIirCu<T>* f_segment_ptr)
{
    const unsigned short l_idx = threadIdx.x;
    f_segment_ptr[l_idx].m_x = static_cast<T>(0.0);
    f_segment_ptr[l_idx].m_y = static_cast<T>(0.0);
    f_segment_ptr[l_idx].m_t1_vec[0] = static_cast<T>(0.0);
    f_segment_ptr[l_idx].m_t1_vec[1] = static_cast<T>(0.0);
    f_segment_ptr[l_idx].m_t2_vec[0] = static_cast<T>(0.0);
    f_segment_ptr[l_idx].m_t2_vec[1] = static_cast<T>(0.0);
}

template <typename T>
CIirCu<T>::CIirCu()
{
    m_x = static_cast<T>(0.0);
    m_y = static_cast<T>(0.0);
    m_t1_vec[0] = static_cast<T>(0.0);
    m_t1_vec[1] = static_cast<T>(0.0);
    m_t2_vec[0] = static_cast<T>(0.0);
    m_t2_vec[1] = static_cast<T>(0.0);
}

template <typename T>
CIirCu<T>::CIirCu(const CIirCoeffs<T>& f_coeffs)
{
    m_x = static_cast<T>(0.0);
    m_y = static_cast<T>(0.0);
    m_t1_vec[0] = static_cast<T>(0.0);
    m_t1_vec[1] = static_cast<T>(0.0);
    m_t2_vec[0] = static_cast<T>(0.0);
    m_t2_vec[1] = static_cast<T>(0.0);
    m_coeffs = f_coeffs;
}

template <typename T>
CIirFilterCu<T>::CIirFilterCu()
{
}

template <typename T>
CIirFilterCu<T>::~CIirFilterCu()
{
}

template <typename T>
T CIirFilterCu<T>::doFiltering(T f_inputVal)
{
    T l_outputVal;
    dim3 l_fwd_grid(1, 1, 1);
    dim3 l_fwd_thr(1, 1, 1);
    fwdFiltering<T> << < l_fwd_grid, l_fwd_thr >> > (m_iirSegments_vec, m_numOfStages, m_bitShifter, f_inputVal, m_n, m_outputVal_ptr);
    hipMemcpy(&l_outputVal, m_outputVal_ptr, sizeof(T), hipMemcpyDeviceToHost);
    return l_outputVal;
}

template <typename T>
void CIirFilterCu<T>::doRwdFiltering()
{
    dim3 l_rwd_grid(1, 1, 1);
    dim3 l_rwd_thr(m_numOfStages, 1, 1);
    rwdFiltering<T> << < l_rwd_grid, l_rwd_thr >> > (m_iirSegments_vec, m_n);
    m_n++;
    m_n &= 1;
}

template <typename T>
void CIirFilterCu<T>::resetFilter()
{
    m_n = 0;
    dim3 l_res_grid(1, 1, 1);
    dim3 l_res_thr(m_numOfStages, 1, 1);
    resetSegment<T> << < l_res_grid, l_res_thr >> > (m_iirSegments_vec);
}

template <typename T>
void CIirFilterCu<T>::destroyFilter()
{
    hipFree(m_iirSegments_vec);
    m_iirSegments_vec = 0;
    hipFree(m_outputVal_ptr);
    m_outputVal_ptr = 0;
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunction_i16()
{
    CIirFilterCu<short> l_tempObj;
    l_tempObj.doFiltering(0);
    l_tempObj.doRwdFiltering();
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
    CIirCoeffs<short> l_tempCoeffs;
    CIirCu<short> l_tempIir_1;
    CIirCu<short> l_tempIir_2(l_tempCoeffs);
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunction_i32()
{
    CIirFilterCu<long> l_tempObj;
    l_tempObj.doFiltering(0);
    l_tempObj.doRwdFiltering();
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
    CIirCoeffs<long> l_tempCoeffs;
    CIirCu<long> l_tempIir_1;
    CIirCu<long> l_tempIir_2(l_tempCoeffs);
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunction_i64()
{
    CIirFilterCu<long long> l_tempObj;
    l_tempObj.doFiltering(0);
    l_tempObj.doRwdFiltering();
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
    CIirCoeffs<long long> l_tempCoeffs;
    CIirCu<long long> l_tempIir_1;
    CIirCu<long long> l_tempIir_2(l_tempCoeffs);
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunction_f32()
{
    CIirFilterCu<float> l_tempObj;
    l_tempObj.doFiltering(0);
    l_tempObj.doRwdFiltering();
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
    CIirCoeffs<float> l_tempCoeffs;
    CIirCu<float> l_tempIir_1;
    CIirCu<float> l_tempIir_2(l_tempCoeffs);
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunction_f64()
{
    CIirFilterCu<double> l_tempObj;
    l_tempObj.doFiltering(0);
    l_tempObj.doRwdFiltering();
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
    CIirCoeffs<double> l_tempCoeffs;
    CIirCu<double> l_tempIir_1;
    CIirCu<double> l_tempIir_2(l_tempCoeffs);
}


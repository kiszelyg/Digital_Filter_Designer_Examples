#include "hip/hip_runtime.h"
// fircu_cu.txt template file, version: 01_01_01
// GENERATED FILE! MODIFY THIS FILE ONLY AT YOUR OWN RESPONSIBLITY!
// An identical behaviour to the simulation results can be assured only if this file remains unchanged!
// Code file of a general CUDA (R) FIR filter implementation

#include "fircu.cuh"

template<typename T>
__global__ void sumVector(T* f_data_ptr)
{
    const unsigned short l_threadId = threadIdx.x;
    unsigned short l_idxStep = 1;
    unsigned short l_threadNr = blockDim.x;
    while (0 < l_threadNr)
    {
        if (l_threadId < (l_threadNr))
        {
            f_data_ptr[(2 * l_threadId * l_idxStep)] += f_data_ptr[((2 * l_threadId + 1) * l_idxStep)];
        }
        l_idxStep <<= 1;
        l_threadNr >>= 1;
        __syncthreads();
    }
}

template<typename T>
__global__ void mulVectors(unsigned short f_offset, const T* f_coeff_ptr, const T* f_buffer_ptr, T* f_res_ptr)
{
    const unsigned short l_threadId = threadIdx.x;
    const unsigned short l_idx = (f_offset - l_threadId) < 0 ? (f_offset - l_threadId + blockDim.x) : (f_offset - l_threadId);
    f_res_ptr[l_threadId] = f_coeff_ptr[l_threadId] * f_buffer_ptr[l_idx];
}

template<typename T>
__global__ void resetBuffer(T* f_buffer_ptr)
{
    const unsigned short l_idx = threadIdx.x;
    f_buffer_ptr[l_idx] = static_cast<T>(0.0);
}

template <typename T>
CFirCu<T>::CFirCu()
{
    m_ptrOffset = 0;
}

template <typename T>
CFirCu<T>::~CFirCu()
{
    if (m_coeffs_vec != 0)
    {
        hipFree(m_coeffs_vec);
        m_coeffs_vec = 0;
    }
    if (m_inputBuffer_vec != 0)
    {
        hipFree(m_inputBuffer_vec);
        m_inputBuffer_vec = 0;
    }
    if (m_res_vec != 0)
    {
        hipFree(m_res_vec);
        m_res_vec = 0;
    }
}

template <typename T>
T CFirCu<T>::doFiltering(T f_inputVal)
{
    T l_outputVal = 0;
    dim3 l_mul_grid(1, 1, 1);
    dim3 l_mul_thr(m_numOfStages, 1, 1);
    hipMemcpy(&m_inputBuffer_vec[m_ptrOffset], &f_inputVal, sizeof(T), hipMemcpyHostToDevice);
    mulVectors<T> << < l_mul_grid, l_mul_thr >> > (m_ptrOffset, m_coeffs_vec, m_inputBuffer_vec, m_res_vec);
    dim3 l_sum_grid(1, 1, 1);
    dim3 l_sum_thr(m_resSize >> 1, 1, 1);
    sumVector<T> << < l_sum_grid, l_sum_thr >> > (m_res_vec);
    hipMemcpy(&l_outputVal, m_res_vec, sizeof(T), hipMemcpyDeviceToHost);
    m_ptrOffset = (m_ptrOffset < (m_numOfStages - 1)) ? ++m_ptrOffset : 0;
    return l_outputVal;
}

template <typename T>
void CFirCu<T>::resetFilter()
{
    m_ptrOffset = 0;
    dim3 l_resetBuffer_grid(1, 1, 1);
    dim3 l_resetBuffer_thr(m_numOfStages, 1, 1);
    resetBuffer<T> << < l_resetBuffer_grid, l_resetBuffer_thr >> > (m_inputBuffer_vec);
}

template <typename T>
long CFirCu<T>::destroyFilter()
{
    long l_retVal = RETURN_OK;
    hipError_t l_cudaRetVal = hipSuccess;
    l_cudaRetVal = (hipFree(m_coeffs_vec));
    if (hipSuccess != l_cudaRetVal)
    {
        printf("FAILURE! Releasing CUDA memory for the results is not possible! Error: %d\n", l_cudaRetVal);
        l_retVal = RETURN_ERROR;
    }
    m_coeffs_vec = 0;
    l_cudaRetVal = (hipFree(m_inputBuffer_vec));
    if (hipSuccess != l_cudaRetVal)
    {
        printf("FAILURE! Releasing CUDA memory for the coefficients is not possible! Error: %d\n", l_cudaRetVal);
        l_retVal = RETURN_ERROR;
    }
    m_inputBuffer_vec = 0;
    l_cudaRetVal = (hipFree(m_res_vec));
    if (hipSuccess != l_cudaRetVal)
    {
        printf("FAILURE! Releasing CUDA memory for the input buffer is not possible! Error: %d\n", l_cudaRetVal);
        l_retVal = RETURN_ERROR;
    }
    m_res_vec = 0;
    return(l_retVal);
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunctionDouble()
{
    CFirCu<double> l_tempObj;
    l_tempObj.doFiltering(0.0);
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunctionFloat()
{
    CFirCu<float> l_tempObj;
    l_tempObj.doFiltering(0.0);
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunctionLongLong()
{
    CFirCu<long long> l_tempObj;
    l_tempObj.doFiltering(0);
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunctionLong()
{
    CFirCu<long> l_tempObj;
    l_tempObj.doFiltering(0);
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
}

// Do NOT call this function, only to avoid the linker error
void TemporaryFunctionShort()
{
    CFirCu<short> l_tempObj;
    l_tempObj.doFiltering(0);
    l_tempObj.resetFilter();
    l_tempObj.destroyFilter();
}

